#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"
#include ""

using namespace std;

template<typename T>
void printVector(const T* a, int size)
{
	for (int i = 0; i < size; i++)
		cout << setw(3) << a[i];
	cout << endl;
}

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	// int i = threadIdx.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		c[i] = a[i] + b[i];

	// printf("ThreadIdx(% u, % u, % u)\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	const int threadsPerBlock = 256; // 최대 deviceProp.maxThreadsPerBlock = 1024 까지 가능

	const int size = 1024 * 1024 * 256; // 여기서는 블럭을 여러 개 사용해야 하는 큰 size
	//const int size = 37;

	// 생각해볼 점: 블럭이 몇 개가 필요할까?

	//vector<int> a(size);
	//vector<int> b(size);
	//vector<int> c_single(size);     // 결과 확인용
	//vector<int> c(size, -1); // CUDA에서 계산한 결과 저장

	int* a = nullptr;
	int* b = nullptr;
	int* c_single = nullptr;
	int* c = nullptr;

	hipHostMalloc(&a, sizeof(int) * size); // pinned-memory
	hipHostMalloc(&b, sizeof(int) * size);
	hipHostMalloc(&c_single, sizeof(int) * size); // GPU 통신에 사용되지 않기 때문에 꼭 pinned-memory를 사용할 필요는 없음
	hipHostMalloc(&c, sizeof(int) * size);

	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
		c_single[i] = a[i] + b[i];
	}

	cout << "Add vectors using CUDA" << endl;

	{

		// 참고: hipStreamSynchronize()를 사용하고 싶은 경우
		//{
		//	hipSetDeviceFlags(hipDeviceScheduleBlockingSync);

		//	unsigned int flags; hipError_t err = hipGetDeviceFlags(&flags);
		//	if (err != hipSuccess) {
		//		cerr << "Failed to get device flags: " << hipGetErrorString(err) << endl; return 1;
		//	}
		//	if (flags & hipDeviceScheduleBlockingSync) {
		//		cout << "hipDeviceScheduleBlockingSync: set" << endl;
		//	}
		//	else {
		//		cout << "hipDeviceScheduleBlockingSync: NOT set" << endl;
		//	}
		//}

		hipStream_t stream;
		hipStreamCreate(&stream);

		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_a, size * sizeof(int)); // input a
		hipMalloc((void**)&dev_b, size * sizeof(int)); // input b
		hipMalloc((void**)&dev_c, size * sizeof(int)); // output c

		hipEvent_t start, stop;// 시간 측정을 위한 CUDA 이벤트 생성 (시간측정도 Nsight로 할 수 있습니다.)
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0); // 시작 시간 기록 (H2D -> Kernel -> D2H)

		hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream); // 비동기적으로 복사 복사
		hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream);

		int blocks = int(ceil(float(size) / threadsPerBlock)); // 블럭 여러 개 사용
		addKernel << <blocks, threadsPerBlock, 0, stream >> > (dev_a, dev_b, dev_c, size);

		// 결과 복사 device -> host
		hipMemcpyAsync(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost, stream);

		hipEventRecord(stop, 0);  // 끝나는 시간 기록
		hipDeviceSynchronize();       // kernel이 끝날때까지 대기 (동기화)

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
		cout << "Time elapsed: " << milliseconds << " ms" << endl;

		// 안내: kernel 실행 후 hipGetLastError() 생략

		// 결과 확인
		if (size < 40) { // size가 작을 경우에는 출력해서 확인
			printVector(a, size);
			printVector(b, size);
			printVector(c_single, size);
			printVector(c, size);
		}

		for (int i = 0; i < size; i++)
			if (c_single[i] != c[i])
			{
				cout << "Wrong result" << endl;
				return 1;
			}

		cout << "Correct" << endl;

		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c_single);
		hipHostFree(c);

		hipDeviceReset();
	}



	return 0;
}

