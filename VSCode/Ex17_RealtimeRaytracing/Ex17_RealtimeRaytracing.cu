﻿// CUDA 공식 예제를 변경하였습니다.

#include "RealtimeRaytracing.h"

#include "hip/hip_runtime_api.h" // copied from cuda_sample

int main(int argc, char **argv) {

    RealtimeRaytracing app;

    std::string image_filename = "image.jpg";

    try {
        // This app only works on ppm images
        app.LoadImageData(image_filename);
        app.Run();
    } catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
