﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <chrono>
#include <functional>
#include <string>

using namespace std;

// 참고 자료
// - https://github.com/umfranzw/cuda-reduction-example/tree/master/reduce0
// - https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

void timedRun(const string name, const function<void()> &func) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    auto startCpu = chrono::high_resolution_clock::now(); // CPU 시간측정 시작
    hipEventRecord(start, 0);                            // GPU 시간측정 시작

    func(); // 실행

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);                         // GPU 시간측정 종료
    auto endCpu = chrono::high_resolution_clock::now(); // CPU 시간측정 종료

    float elapsedGpu = 0;
    hipEventElapsedTime(&elapsedGpu, start, stop);
    chrono::duration<float, milli> elapsedCpu = endCpu - startCpu;
    cout << name << ": CPU " << elapsedCpu.count() << " ms, GPU " << elapsedGpu << "ms" << endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// 교재: "Programming Massively Parallel Processors: A Hands-on Approach" 4th

__global__ void atomicSumReductionKernel(float *input, float *output) {

    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

    output[0] += input[0];

    // TODO; // <- AtomicAdd()
}

__global__ void convergentSumReductionKernel(float *input,
                                             float *output) { // block 하나로 처리가능한 크기
    unsigned int i = threadIdx.x;

    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
        if (threadIdx.x < stride) {
            input[i] += input[i + stride];
        }
        __syncthreads(); // <- 같은 블럭 안에 있는 쓰레드들 동기화
    }
    if (threadIdx.x == 0)
        *output = input[0];
}

__global__ void sharedMemorySumReductionKernel(float *input, float *output) {

    extern __shared__ float inputShared[]; // <- 블럭 안에서 여러 쓰레드들이 공유하는 빠른 메모리

    unsigned int t = threadIdx.x;

    inputShared[t] = input[t] + input[t + blockDim.x];

    for (unsigned int stride = blockDim.x / 2; stride >= 1; stride /= 2) {

        __syncthreads();

        if (threadIdx.x < stride) {
            inputShared[t] += inputShared[t + stride];
        }
    }
    if (t == 0)
        *output = inputShared[0];
}

__global__ void segmentedSumReductionKernel(float *input, float *output) {
    extern __shared__ float inputShared[];

    unsigned int segment = 2 * blockDim.x * blockIdx.x;
    unsigned int i = segment + threadIdx.x;
    unsigned int t = threadIdx.x;

    // TODO: 위의 두 개를 잘 합치면 됩니다.
}

int main(int argc, char *argv[]) {

    const int size = 1024 * 1024 * 32;

    // 배열 만들기
    vector<float> arr(size);
    srand(uint32_t(time(nullptr)));
    for (int i = 0; i < size; i++)
        arr[i] = (float)rand() / RAND_MAX;

    // CPU에서 합 구하기
    float sumCpu = 0.0f;
    timedRun("CPU Sum", [&]() {
        for (int i = 0; i < size; i++) {
            sumCpu += arr[i];
        }
    });

    // GPU 준비
    float *dev_input;
    float *dev_output;

    int threadsPerBlock = 1024;

    hipMalloc(&dev_input, size * sizeof(float));
    hipMalloc(&dev_output, sizeof(float));
    hipMemcpy(dev_input, arr.data(), size * sizeof(float), hipMemcpyHostToDevice);

     timedRun("Atomic", [&]() {
         int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
         atomicSumReductionKernel<<<numBlocks, threadsPerBlock>>>(dev_input, dev_output);
     }); // 68 ms

    // timedRun("GPU Sum", [&]() {
    //     convergentSumReductionKernel<<<1, threadsPerBlock>>>(dev_input, dev_output); // 블럭이
    //     하나일 때만 사용
    // });

    // timedRun("GPU Sum", [&]() {
    //     sharedMemorySumReductionKernel<<<1, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(
    //         dev_input, dev_output); // 블럭이 하나일 때만 사용
    // });

    // timedRun("Segmented", [&]() {
    //     int numBlocks = (size / 2 + threadsPerBlock - 1) / threadsPerBlock; // size 나누기 2 주의
    //     segmentedSumReductionKernel<<<numBlocks, threadsPerBlock,
    //                                   threadsPerBlock * sizeof(float)>>>(dev_input, dev_output);
    // });  // 1 ms 근처

    float sumGpu = 0.0f;
    hipMemcpy(&sumGpu, dev_output, sizeof(float), hipMemcpyDeviceToHost); // 숫자 하나만 복사

    cout << "sumCpu = " << sumCpu << ", sumGpu = " << sumGpu << endl;
    cout << "Avg Error = " << std::abs((sumCpu - sumGpu)) / size << endl;

    hipFree(dev_input);
    hipFree(dev_output);

    return EXIT_SUCCESS;
}
